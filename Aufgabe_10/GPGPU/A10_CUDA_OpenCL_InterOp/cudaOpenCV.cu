#include "hip/hip_runtime.h"
#include "cudaKernel.h"
#ifdef WIN32
#include <windows.h>
#endif
#include "hip/hip_runtime.h"
#include "cuda_gl_interop.h"
#include "opencv2/opencv.hpp"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CHANNELS 3
#define SOBEL_RADIUS 1
#define HISTOGRAMMSIZE 256

using namespace cv;

hipGraphicsResource_t vboRes; // cuda vertex buffer reference
hipGraphicsResource_t texRes; // cuda texture reference
hipGraphicsResource_t texResGray; // cuda texture reference
int *d_width, *d_height; // device memory var�ables
VideoCapture cap("C:/Users/sbenz/Desktop/OpenCVReadVideo/Videos/robotica_1080.mp4"); // Opencv video capture
Mat currFrame;


__global__ void getHistogrammKernel(int cu_image_width, int cu_image_height, unsigned char *cu_src_image, unsigned int *cu_dest_histogramm)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows


	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;

	while (x < cu_image_width && y < cu_image_height) {
		int index = y * cu_image_width + x;

		atomicAdd(&(cu_dest_histogramm[cu_src_image[index]]), 1);

		x += stride_x;
		y += stride_y;
	}
}

__global__ void sobelFilterKernel(int cu_image_width, int cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows

												   //Calc index
	int global_index = (y * cu_image_width)+ x;

	if (x >= SOBEL_RADIUS && x < cu_image_width - 1 && y >= SOBEL_RADIUS && y < cu_image_height - 1) {
		//Calc Sobel X & Y if the thread is inside the filter area
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;

		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				sobel_gradient_x += cu_src_image[(y + j) * cu_image_width + (x + k)] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += cu_src_image[(y + j) * cu_image_width + (x + k)] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}

		//Calc Sobel magnitude and save it to the image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));

		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
	else {
		if (x < cu_image_width && y < cu_image_height) {
			cu_dest_image[global_index] = 0;
		}
	}
}

//Kernel rgb to grayscale function
__global__ void rgbToGrayscaleKernel(int cu_image_width, int cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b, gray;

	if (x < cu_image_width && y < cu_image_height) {
		int grayOffset = (y * (cu_image_width) + x);
		int rgbOffset = grayOffset * CHANNELS;

		b = cu_src_image[rgbOffset];
		g = cu_src_image[rgbOffset + 1];
		r = cu_src_image[rgbOffset + 2];

		gray = 0.21 * r + 0.71 *g + 0.07 *b;

		cu_dest_image[grayOffset] = gray;
	}
}

__global__ void colorKernel(int cu_image_width, int cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b, gray;

	if (x < cu_image_width && y < cu_image_height) {
		int grayOffset = (y * (cu_image_width) + x);
		int rgbOffset = grayOffset * CHANNELS;
		int rgbaOffset = grayOffset * (CHANNELS + 1);

		b = cu_src_image[rgbOffset];
		g = cu_src_image[rgbOffset + 1];
		r = cu_src_image[rgbOffset + 2];
		
		cu_dest_image[rgbaOffset] = r;
		cu_dest_image[rgbaOffset + 1] = g;
		cu_dest_image[rgbaOffset + 2] = b;
		cu_dest_image[rgbaOffset + 3] = 0;
	}
}

void cudaGetOpenCVImageSize(unsigned int &cols, unsigned int &rows) {
	cols = 1440;
	rows = 1080;
}

void cudaInit ( unsigned int texId, unsigned int texIdGray, unsigned int vboId, unsigned int cols, unsigned int rows){
	
	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Register Gl Buffer
	err = hipGraphicsGLRegisterBuffer(&vboRes, vboId, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Register Gl Texture
	err = hipGraphicsGLRegisterImage(&texRes, texId, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsGLRegisterImage(&texResGray, texIdGray, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Init device memory
	err = hipMalloc((void **)&d_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_width, &cols, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_height, &rows, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

}

void cudaClearAllocatedData() {
	hipFree(d_width);
	hipFree(d_height);
}

void copyColorImage(unsigned char *src_image, int width, int height) {
	unsigned char *d_dest_image, *d_src_image;
	hipArray *texArray;

	unsigned int imgSize = width * height * sizeof(unsigned char);
	unsigned int imgSizeRgb = imgSize * CHANNELS;
	unsigned int imgSizeRgba = imgSize * (CHANNELS + 1);

	hipError_t err = hipSuccess;

	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSizeRgb);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSizeRgb, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_dest_image, imgSizeRgba);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Texture
	err = hipGraphicsMapResources(1, &texRes);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsSubResourceGetMappedArray(&texArray, texRes, 0, 0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((width - 1) / threads + 1, (height - 1) / threads + 1, 1);

	colorKernel <<<blocks_per_grid, threads_per_block >> >(width, height, d_src_image, d_dest_image);

	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToArray(texArray, 0, 0, d_dest_image, imgSizeRgba, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsUnmapResources(1, &texRes);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_dest_image);
	hipFree(d_src_image);
	hipFree(d_height);
	hipFree(d_width);
}

void rgbToGrayscale(unsigned char *src_image, int width, int height) {
	unsigned char *d_src_image;
	unsigned char *d_dest_image;
	hipArray *texArray;

	unsigned int imgSize = width * height * sizeof(unsigned char);
	unsigned int imgSizeRgb = imgSize * CHANNELS;

	hipError_t err = hipSuccess;

	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSizeRgb);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSizeRgb, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Texture
	err = hipGraphicsMapResources(1, &texResGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsSubResourceGetMappedArray(&texArray, texResGray, 0, 0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((width - 1) / threads + 1, (height - 1) / threads + 1, 1);

	rgbToGrayscaleKernel<<<blocks_per_grid, threads_per_block>>>(width, height, d_src_image, d_dest_image);

	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToArray(texArray, 0, 0, d_dest_image, imgSize, hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsUnmapResources(1, &texResGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_dest_image);
	hipFree(d_src_image);
}

void sobelFilter(unsigned char *src_image, int width, int height)
{
	unsigned char *d_src_image, *d_dest_image_gray, *d_dest_image_sobel;
	hipArray *texArray;

	unsigned int imgSize = width * height * sizeof(unsigned char);
	unsigned int imgSizeRgb = imgSize * CHANNELS;

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSizeRgb);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSizeRgb, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image_gray, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_dest_image_sobel, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Texture
	err = hipGraphicsMapResources(1, &texResGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsSubResourceGetMappedArray(&texArray, texResGray, 0, 0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Per Grid N/16 Blocks
	dim3 blocks_per_grid((width - 1) / threads + 1, (height - 1) / threads + 1, 1);

	rgbToGrayscaleKernel <<<blocks_per_grid, threads_per_block >> >(width, height, d_src_image, d_dest_image_gray);

	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	sobelFilterKernel <<<blocks_per_grid, threads_per_block>>>(width, height, d_dest_image_gray, d_dest_image_sobel);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyToArray(texArray, 0, 0, d_dest_image_sobel, imgSize, hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipGraphicsUnmapResources(1, &texResGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_src_image);
	hipFree(d_dest_image_gray);
	hipFree(d_dest_image_sobel);
};

void getHistogramm(int width, int height, unsigned char *src_image)
{
	unsigned char *d_src_image;
	unsigned int  *d_dest_histogramm;
	float *vboPtr;

	unsigned int imgSize = (width * height) * sizeof(unsigned char);

	unsigned int histogrammSize = HISTOGRAMMSIZE * sizeof(unsigned int);

	size_t vboSize = HISTOGRAMMSIZE * 3 * sizeof(float);


	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_histogramm, histogrammSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//VBO
	err = hipGraphicsResourceGetMappedPointer((void **)&vboPtr, &vboSize, vboRes);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Per Grid N/16 Blocks
	dim3 blocks_per_grid((width - 1) / threads + 1, (height - 1) / threads + 1, 1);
	getHistogrammKernel << <blocks_per_grid, threads_per_block >> >(width, height, d_src_image, d_dest_histogramm);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}


	hipFree(d_dest_histogramm);
	hipFree(d_src_image);
};


void displayGrayscaleImage(void) {
	rgbToGrayscale(currFrame.data, currFrame.cols, currFrame.rows);
}

void applySobelFilter(void) {
	sobelFilter(currFrame.data, currFrame.cols, currFrame.rows);
}

void displayColorImage(void) {
	copyColorImage(currFrame.data, currFrame.cols, currFrame.rows);
}

void displayHistogramm(void) {

}

int cudaExecOneStep(void) {
	// Check if videocapture suceeded
	if (!cap.isOpened()) {
		return -1;
	}

	cap >> currFrame;

	// Check if the video is done
	if (currFrame.empty()) {
		cap.set(CV_CAP_PROP_POS_FRAMES, 0);
	}

	return 0;
}