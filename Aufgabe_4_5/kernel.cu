#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "fix.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <cstdlib>
#include <algorithm>
//
#define CHANNELS 3
#define REDCHANNEL 'r'
#define GREENCHANNEL 'g'
#define BLUECHANNEL 'b'
#define GRAYSCLAEREDCHANNEL 0.21
#define GRAYSCLAEGREENCHANNEL 0.71
#define GRAYSCLAEBLUECHANNEL 0.07
#define SOBEL_RADIUS 1
#define TILE_W 16
#define BLOCK_W (TILE_W + 2*SOBEL_RADIUS)

//Kernel sobel function
__global__ void sobelFilterKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{

	__shared__ char ds_Img[BLOCK_W][BLOCK_W];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = bx * TILE_W + tx -SOBEL_RADIUS; //cols
	int y = by * TILE_W + ty -SOBEL_RADIUS; //rows

	//Make sure x/y are not negative
	if (x < 0) {
		x = 0;
	}

	if (y < 0) {
		y = 0;
	}

	//Calc index of global memory
	int global_index = (y * (*cu_image_width) + x);

	//Load Data into Shared Memory
	//Insert 0 if the thread is supposed to fill the filter radius border of the tile
	if (x >= 0 && x < *cu_image_width - 1 && y >=  0 && y < *cu_image_height - 1) {
		ds_Img[ty][tx] = cu_src_image[global_index];
	}
	else {
		ds_Img[ty][tx] = 0;
	}
	__syncthreads();

	//Calc Sobel X & Y if the thread is inside the filter area
	if ((tx >= SOBEL_RADIUS) && (tx <= TILE_W) &&
		(ty >= SOBEL_RADIUS) && (ty <= TILE_W)){
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;
		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				sobel_gradient_x += ds_Img[ty + j][tx + k] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += ds_Img[ty + j][tx + k] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}
		//Calc Sobel magnitude and save it to the original image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));
		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
}

//Kernel rgb to grayscale function
__global__ void rgbToGrayscaleKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b, gray;

	if (x < *cu_image_width && y < *cu_image_height) {
		int grayOffset = (y * (*cu_image_width) + x);
		int rgbOffset = grayOffset * CHANNELS;

		b = cu_src_image[rgbOffset];
		g = cu_src_image[rgbOffset + 1];
		r = cu_src_image[rgbOffset + 2];

		gray = 0.21 * r + 0.71 *g + 0.07 *b;

		cu_dest_image[grayOffset] = gray;
	}
}

//Kernel ColorChannel function
__global__ void setColorChannelKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image, unsigned char *cu_channel_to_keep)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b;

	if (x < *cu_image_width && y < *cu_image_height) {
		int offset = (y * (*cu_image_width) + x) * CHANNELS;

		switch (*cu_channel_to_keep)
		{
		case BLUECHANNEL:
			b = cu_src_image[offset];
			g = 0;
			r = 0;
			break;
		case GREENCHANNEL:
			b = 0;
			g = cu_src_image[offset + 1];
			r = 0;
			break;
		case REDCHANNEL:
			b = 0;
			g = 0;
			r = cu_src_image[offset + 2];
			break;
		default: //Defaults to REDCHANNEL
			b = 0;
			g = 0;
			r = cu_src_image[offset + 2];
			break;
		}

		cu_dest_image[offset] = b; //B
		cu_dest_image[offset + 1] = g; //G
		cu_dest_image[offset + 2] = r; //R
	}

};

void setColorChannel(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image, unsigned char channel_to_keep)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image, *d_channel_to_keep;

	unsigned int imgSize = (image_width * image_height) * CHANNELS * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy channel to keep to gpu
	err = hipMalloc((void **)&d_channel_to_keep, sizeof(unsigned char));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_channel_to_keep, &channel_to_keep, sizeof(unsigned char), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_dest_image, dest_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	double const threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	setColorChannelKernel << <blocks_per_grid, threads_per_block >> >(d_image_width, d_image_height, d_src_image, d_dest_image, d_channel_to_keep);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_channel_to_keep);
	hipFree(d_src_image);
	hipFree(d_dest_image);
}


void rgbToGrayscale(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image;

	unsigned int imgSizeRgb = (image_width * image_height) * CHANNELS * sizeof(unsigned char);
	unsigned int imgSizeGray = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSizeRgb);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSizeRgb, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSizeGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_image, dest_image, imgSizeGray, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	double const threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	rgbToGrayscaleKernel << <blocks_per_grid, threads_per_block >> >(d_image_width, d_image_height, d_src_image, d_dest_image);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSizeGray, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
	hipFree(d_dest_image);
};

void sobelFilter(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image;

	unsigned int imgSize = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_image, dest_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	double const threads = BLOCK_W;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((image_width -1) / TILE_W + 1, (image_height - 1) / TILE_W + 1, 1);
	sobelFilterKernel << <blocks_per_grid, threads_per_block >> >(d_image_width, d_image_height, d_src_image, d_dest_image);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
	hipFree(d_dest_image);
};