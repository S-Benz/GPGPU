#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "fix.h"
#include ""
#include <hip/device_functions.h>
#include <device_atomic_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <math.h>
#include <cstdlib>
//
#define CHANNELS 3
#define REDCHANNEL 'r'
#define GREENCHANNEL 'g'
#define BLUECHANNEL 'b'
#define GRAYSCLAEREDCHANNEL 0.21
#define GRAYSCLAEGREENCHANNEL 0.71
#define GRAYSCLAEBLUECHANNEL 0.07
#define SOBEL_RADIUS 1
#define TILE_W 16
#define BLOCK_W (TILE_W + 2*SOBEL_RADIUS)
#define ANGLE 50
#define HISTOGRAMMSIZE 256

__global__ void sobelFilterKernelTiledStreams(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image, int sobel_offset)
{
	__shared__ char ds_Img[BLOCK_W][BLOCK_W];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = bx * TILE_W + tx - SOBEL_RADIUS; //cols
	int y = by * TILE_W + ty - SOBEL_RADIUS; //rows

											 //Make sure x/y are not negative
	if (x < 0) {
		x = 0;
	}

	if (y < 0) {
		y = 0;
	}

	//Calc index of global memory
	int global_index = sobel_offset + (y * (*cu_image_width) + x);

	//Load Data into Shared Memory
	//Insert 0 if the thread is supposed to fill the filter radius border of the tile
	if (x >= 0 && x < *cu_image_width - 1 && y >= 0 && y < *cu_image_height - 1) {
		ds_Img[ty][tx] = cu_src_image[global_index];
	}
	else {
		if (x < *cu_image_width && y < *cu_image_height) {
			ds_Img[ty][tx] = 0;
		}
	}
	__syncthreads();

	//Calc Sobel X & Y if the thread is inside the filter area
	if ((tx >= SOBEL_RADIUS) && (tx <= TILE_W) &&
		(ty >= SOBEL_RADIUS) && (ty <= TILE_W)) {
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;
		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				sobel_gradient_x += ds_Img[ty + j][tx + k] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += ds_Img[ty + j][tx + k] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}
		//Calc Sobel magnitude and save it to the original image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));
		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
}

__global__ void sobelFilterKernelStreams(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image, int offset_sobel)
{
	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows

												   //Calc index
	int global_index = offset_sobel + (y * (*cu_image_width) + x);

	if (x >= SOBEL_RADIUS && x < *cu_image_width - 1 && y >= SOBEL_RADIUS && y < *cu_image_height - 1) {
		//Calc Sobel X & Y if the thread is inside the filter area
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;

		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				int sobel_index = offset_sobel + (y + j) * (*cu_image_width) + (x + k);
				sobel_gradient_x += cu_src_image[sobel_index] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += cu_src_image[sobel_index] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}

		//Calc Sobel magnitude and save it to the image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));

		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
	else {
		if (x < *cu_image_width && y < *cu_image_height) {
			cu_dest_image[global_index] = 0;
		}
	}
}

//Kernel rgb to grayscale function with streams
__global__ void rgbToGrayscaleKernelStream(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image, int offset_rgb, int offset_gray)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b, gray;

	if (x < *cu_image_width && y < *cu_image_height) {
		int offset = (y * (*cu_image_width) + x);
		int grayOffset =  offset + offset_gray;
		int rgbOffset = offset_rgb + offset  * CHANNELS;

		b = cu_src_image[rgbOffset];
		g = cu_src_image[rgbOffset + 1];
		r = cu_src_image[rgbOffset + 2];

		gray = 0.21 * r + 0.71 *g + 0.07 *b;

		cu_dest_image[grayOffset] = gray;
	}
}

__global__ void getHistogrammTiledKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned int *cu_dest_histogramm) {

	__shared__ unsigned int smem[HISTOGRAMMSIZE];

	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows

	int shared_index = threadIdx.x + threadIdx.y * blockDim.x;
	int stride_shared = blockDim.x * blockDim.y;

	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;

	//Init shared memory histogramm with 0's
	for (int i = shared_index; i < HISTOGRAMMSIZE; i += stride_shared) {
		smem[i] = 0;
	}

	__syncthreads();

	//Add data to histogramm in shared memory
	while (x < *cu_image_width && y < *cu_image_height) {
		int index = y * *cu_image_width + x;

		atomicAdd(&(smem[cu_src_image[index]]), 1);

		x += stride_x;
		y += stride_y;
	
	}

	__syncthreads();
	/*
	long test = 0;
	if (threadIdx.x == 0) {
		for (int j = 0; j < HISTOGRAMMSIZE; j++) {
			test += smem[j];
		}
		printf("Smem total per block: %lu \n", test);
	}*/
	
	//Add shared memory histogramm part to global memory histogramm
	for (int i = shared_index; i < HISTOGRAMMSIZE; i+= stride_shared) {
		atomicAdd(&(cu_dest_histogramm[i]), smem[i]);
	}

	__syncthreads();
	
}

__global__ void getHistogrammKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned int *cu_dest_histogramm)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows


	int stride_x = blockDim.x * gridDim.x;
	int stride_y = blockDim.y * gridDim.y;

	while (x < *cu_image_width && y < *cu_image_height) {
		int index = y * *cu_image_width + x;
		
		atomicAdd(&(cu_dest_histogramm[cu_src_image[index]]), 1);

		x += stride_x;
		y += stride_y;
	}
}

__global__ void sobelFilterTexture(int *cu_image_width, int *cu_image_height, unsigned char *cu_output, hipTextureObject_t cu_texObj, float theta)
{
	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < *cu_image_width - 1 &&  y < *cu_image_height - 1) {
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;

		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				//Calc normalized texture coordinates
				float u = (x + k) / (float)*cu_image_width;
				float v = (y + j) / (float)*cu_image_height;

				// Transform coordinates
				u -= 0.5f;
				v -= 0.5f;

				float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
				float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

				sobel_gradient_x += tex2D<float>(cu_texObj, tu, tv) * 255 * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += tex2D<float>(cu_texObj, tu, tv) * 255 * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}

		//Calc Sobel magnitude and save it to the image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));

		cu_output[y * *cu_image_width + x] = (unsigned char)sobel_magnitude;
	}

};

//Kernel sobel function
__global__ void sobelFilterKernelTiled(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{

	__shared__ char ds_Img[BLOCK_W][BLOCK_W];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;

	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = bx * TILE_W + tx - SOBEL_RADIUS; //cols
	int y = by * TILE_W + ty - SOBEL_RADIUS; //rows

	//Make sure x/y are not negative
	if (x < 0) {
		x = 0;
	}

	if (y < 0) {
		y = 0;
	}

	//Calc index of global memory
	int global_index = (y * (*cu_image_width) + x);

	//Load Data into Shared Memory
	//Insert 0 if the thread is supposed to fill the filter radius border of the tile
	if (x >= 0 && x < *cu_image_width - 1 && y >=  0 && y < *cu_image_height - 1) {
		ds_Img[ty][tx] = cu_src_image[global_index];
	}
	else {
		ds_Img[ty][tx] = 0;
	}
	__syncthreads();

	//Calc Sobel X & Y if the thread is inside the filter area
	if ((tx >= SOBEL_RADIUS) && (tx <= TILE_W) &&
		(ty >= SOBEL_RADIUS) && (ty <= TILE_W)){
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;
		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				sobel_gradient_x += ds_Img[ty + j][tx + k] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += ds_Img[ty + j][tx + k] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}
		//Calc Sobel magnitude and save it to the original image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));
		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
}

__global__ void sobelFilterKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int sobel_x[3][3] = {
		{ 1, 0, -1 },
		{ 2, 0, -2 },
		{ 1, 0, -1 }
	};
	int sobel_y[3][3] = {
		{ 1, 2, 1 },
		{ 0, 0, 0 },
		{ -1, -2, -1 }
	};

	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows

	//Calc index
	int global_index = (y * (*cu_image_width) + x);

	if (x >= SOBEL_RADIUS && x < *cu_image_width - 1 && y >= SOBEL_RADIUS && y < *cu_image_height - 1) {
		//Calc Sobel X & Y if the thread is inside the filter area
		int sobel_gradient_y = 0, sobel_gradient_x = 0, sobel_magnitude = 0;

		for (int j = -SOBEL_RADIUS; j <= SOBEL_RADIUS; j++) {
			for (int k = -SOBEL_RADIUS; k <= SOBEL_RADIUS; k++) {
				sobel_gradient_x += cu_src_image[(y + j) * (*cu_image_width) + (x + k)] * sobel_x[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
				sobel_gradient_y += cu_src_image[(y + j) * (*cu_image_width) + (x + k)] * sobel_y[j + SOBEL_RADIUS][k + SOBEL_RADIUS];
			}
		}

		//Calc Sobel magnitude and save it to the image
		sobel_magnitude = (int)sqrt((float)pow((float)sobel_gradient_x, 2) + (float)pow((float)sobel_gradient_y, 2));

		cu_dest_image[global_index] = (unsigned char)sobel_magnitude;
	}
	else {
		if (x < *cu_image_width && y < *cu_image_height) {
			cu_dest_image[global_index] = 0;
		}
	}
}

//Kernel rgb to grayscale function
__global__ void rgbToGrayscaleKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b, gray;

	if (x < *cu_image_width && y < *cu_image_height) {
		int grayOffset = (y * (*cu_image_width) + x);
		int rgbOffset = grayOffset * CHANNELS;

		b = cu_src_image[rgbOffset];
		g = cu_src_image[rgbOffset + 1];
		r = cu_src_image[rgbOffset + 2];

		gray = 0.21 * r + 0.71 *g + 0.07 *b;

		cu_dest_image[grayOffset] = gray;
	}
}

//Kernel ColorChannel function
__global__ void setColorChannelKernel(int *cu_image_width, int *cu_image_height, unsigned char *cu_src_image, unsigned char *cu_dest_image, unsigned char *cu_channel_to_keep)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //cols
	int y = blockIdx.y * blockDim.y + threadIdx.y; //rows
	unsigned char r, g, b;

	if (x < *cu_image_width && y < *cu_image_height) {
		int offset = (y * (*cu_image_width) + x) * CHANNELS;

		switch (*cu_channel_to_keep)
		{
		case BLUECHANNEL:
			b = cu_src_image[offset];
			g = 0;
			r = 0;
			break;
		case GREENCHANNEL:
			b = 0;
			g = cu_src_image[offset + 1];
			r = 0;
			break;
		case REDCHANNEL:
			b = 0;
			g = 0;
			r = cu_src_image[offset + 2];
			break;
		default: //Defaults to REDCHANNEL
			b = 0;
			g = 0;
			r = cu_src_image[offset + 2];
			break;
		}

		cu_dest_image[offset] = b; //B
		cu_dest_image[offset + 1] = g; //G
		cu_dest_image[offset + 2] = r; //R
	}

};

void setColorChannel(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image, unsigned char channel_to_keep)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image, *d_channel_to_keep;

	unsigned int imgSize = (image_width * image_height) * CHANNELS * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy channel to keep to gpu
	err = hipMalloc((void **)&d_channel_to_keep, sizeof(unsigned char));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_channel_to_keep, &channel_to_keep, sizeof(unsigned char), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_dest_image, dest_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	setColorChannelKernel << <blocks_per_grid, threads_per_block >> >(d_image_width, d_image_height, d_src_image, d_dest_image, d_channel_to_keep);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_channel_to_keep);
	hipFree(d_src_image);
	hipFree(d_dest_image);
}


void rgbToGrayscale(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image;

	unsigned int imgSizeRgb = (image_width * image_height) * CHANNELS * sizeof(unsigned char);
	unsigned int imgSizeGray = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSizeRgb);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSizeRgb, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSizeGray);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_image, dest_image, imgSizeGray, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke, n = Anzahl Threads
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	//rgbToGrayscaleKernel << <blocks_per_grid, threads_per_block >> >(d_image_width, d_image_height, d_src_image, d_dest_image);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSizeGray, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
	hipFree(d_dest_image);
};

void sobelFilter(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image;

	unsigned int imgSize = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_image, dest_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Per Grid N/16 Blocks
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	sobelFilterKernel <<<blocks_per_grid, threads_per_block >>>(d_image_width, d_image_height, d_src_image, d_dest_image);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
	hipFree(d_dest_image);
};

void sobelFilterShared(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image;

	unsigned int imgSize = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_image, dest_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	// Use a Grid with one Block containing Block_width threads
	dim3 threads_per_block_tiled(BLOCK_W, BLOCK_W, 1);
	//Per Grid N/Tile_wisth blocks
	dim3 blocks_per_grid_tiled((image_width - 1) / TILE_W + 1, (image_height - 1) / TILE_W + 1, 1);
	sobelFilterKernelTiled << <blocks_per_grid_tiled, threads_per_block_tiled >> >(d_image_width, d_image_height, d_src_image, d_dest_image);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dest_image, d_dest_image, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
	hipFree(d_dest_image);
};

void sobelFilterTexture(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;

	unsigned int imgSize = (image_width * image_height) * sizeof(unsigned char);

	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Create ChannelDesc
	//Sets output format of the value when the texture is fetched i.e. float texel
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	
	//Create cuda array
	hipArray *cuArray;
	
	//Allocate cuda array
	err = hipMallocArray(&cuArray, &channelDesc, image_width, image_height);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image data to cuda array
	err = hipMemcpyToArray(cuArray, 0, 0, src_image, image_height * image_width * sizeof(unsigned char), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Set Texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	//Set Texture object params
	struct hipTextureDesc textDesc;
	memset(&textDesc, 0, sizeof(textDesc));
	textDesc.addressMode[0] = hipAddressModeMirror;
	textDesc.addressMode[1] = hipAddressModeMirror;
	textDesc.filterMode = hipFilterModeLinear;
	textDesc.readMode = hipReadModeNormalizedFloat;
	textDesc.normalizedCoords = 1;

	//Create Texture Object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &textDesc, NULL);

	unsigned char *output;
	err = hipMalloc(&output, image_height * image_width * sizeof(unsigned char));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	//

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	float angle = 0;
	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Per Grid N/16 Blocks
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	sobelFilterTexture <<<blocks_per_grid, threads_per_block >>>(d_image_width, d_image_height, output, texObj, angle);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dest_image, output, imgSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	hipFree(d_image_width);
	hipFree(d_image_height);
	hipDestroyTextureObject(texObj);
	hipFreeArray(cuArray);
	hipFree(output);
};


void getHistogramm(int image_width, int image_height, unsigned char *src_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image;
	unsigned int  *d_dest_histogramm;

	unsigned int histogramm[HISTOGRAMMSIZE] = { 0 };

	unsigned int imgSize = (image_width * image_height) * sizeof(unsigned char);

	unsigned int histogrammSize = HISTOGRAMMSIZE * sizeof(unsigned int);


	hipError_t err = hipSuccess;

	//Set Device
	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_image_width, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &image_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipMalloc((void **)&d_image_height, sizeof(int));
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &image_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipMalloc((void **)&d_src_image, imgSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_src_image, src_image, imgSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image dest to gpu
	err = hipMalloc((void **)&d_dest_histogramm, histogrammSize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	err = hipMemcpy(d_dest_histogramm, histogramm, histogrammSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}


	unsigned int threads = 16;
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Per Grid N/16 Blocks
	dim3 blocks_per_grid((image_width - 1) / threads + 1, (image_height - 1) / threads + 1, 1);
	getHistogrammTiledKernel <<<blocks_per_grid, threads_per_block >>>(d_image_width, d_image_height, d_src_image, d_dest_histogramm);

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(histogramm, d_dest_histogramm, histogrammSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	long histoCount = 0;

	for (int i = 0; i < HISTOGRAMMSIZE; i++) {
		histoCount += histogramm[i];
	}

	printf("HistogrammSize: %ld \n", histoCount);
	
	hipFree(d_dest_histogramm);
	hipFree(d_image_width);
	hipFree(d_image_height);
	hipFree(d_src_image);
};


void streamAufgabe5(int image_width, int image_height, unsigned char *src_image, unsigned char *dest_image)
{
	int *d_image_width, *d_image_height;
	unsigned char *d_src_image, *d_dest_image, *d_dest_image_sobel;

	unsigned int imgSize = image_width * image_height;
	unsigned int imgSizeRgb = imgSize * CHANNELS * sizeof(unsigned char);
	unsigned int imgSizeGray = imgSize * sizeof(unsigned char);
	
	//Cuda Stream vars
	const unsigned int stream_count = 4;

	//Kernel vars
	unsigned int threads = 16;
	int stream_width = image_width;
	int stream_height = image_height / stream_count;
	int stream_size = stream_width * stream_height;
	int stream_size_gray = stream_size * sizeof(unsigned char);
	int stream_size_rgb = stream_size * CHANNELS * sizeof(unsigned char);

	//tiled sobel
	dim3 threads_per_block_tiled(BLOCK_W, BLOCK_W, 1);
	//Per Grid N/Tile_wisth blocks
	dim3 blocks_per_grid_tiled((stream_width - 1) / TILE_W + 1, (stream_height - 1) / TILE_W + 1, 1);
	
	// Use a Grid with one Block containing 16x16 Threads
	dim3 threads_per_block(threads, threads, 1);
	//Pro Grid N/16 Bl�cke
	dim3 blocks_per_grid((stream_width - 1) / threads + 1, (stream_height - 1) / threads + 1, 1);

	hipStream_t streams[stream_count];

	int dev_count;
	hipDeviceProp_t prop;

	hipError_t err = hipSuccess;

	//Enable device Overlap
	err = hipGetDeviceCount(&dev_count);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Set device to a device with overlap property
	for (int i = 0; i < dev_count; i++) {
		hipGetDeviceProperties(&prop, i);

		if (prop.deviceOverlap) {
			err = hipSetDevice(i);
			if (err != hipSuccess) {
				printf("%s in %s at line %d\n",
					hipGetErrorString(err), __FILE__, __LINE__);
				exit(EXIT_FAILURE);
			}

		}
	}
	
	err = hipHostAlloc((void **)&d_image_width, sizeof(int), hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(d_image_width, &stream_width, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image height to gpu
	err = hipHostAlloc((void **)&d_image_height, sizeof(int), hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_image_height, &stream_height, sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Copy image src to gpu
	err = hipHostAlloc((void **)&d_src_image, imgSizeRgb, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Alloc memory for grayscale image
	err = hipHostAlloc((void **)&d_dest_image, imgSizeGray, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipHostAlloc((void **)&d_dest_image_sobel, imgSizeGray, hipHostMallocDefault);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	//Create cuda Streams & memory for each stream
	for (int i = 0; i < stream_count; i++) {
		//Create cuda Streams
		err = hipStreamCreate(&streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}
		
	}

	//fill memory
	for (int i = 0; i < stream_count; i++) {
		//calc offset for memory copy
		int offset_gray = i * stream_size;
		int offset_rgb = offset_gray * CHANNELS;

		//copy memory for each stream
		err = hipMemcpyAsync(&d_src_image[offset_rgb], &src_image[offset_rgb], stream_size_rgb, hipMemcpyHostToDevice, streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}

		err = hipMemcpyAsync(&d_dest_image[offset_gray], &dest_image[offset_gray], stream_size_gray, hipMemcpyHostToDevice, streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}

		err = hipMemcpyAsync(&d_dest_image_sobel[offset_gray], &dest_image[offset_gray], stream_size_gray, hipMemcpyHostToDevice, streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}


	}

	//execute kernel for grayscale
	for (int i = 0; i < stream_count; i++) {
		int offset_gray = i * stream_size;
		int offset_rgb = offset_gray * CHANNELS;

		rgbToGrayscaleKernelStream<<<blocks_per_grid, threads_per_block, 0, streams[i]>>>(d_image_width, d_image_height, d_src_image, d_dest_image, offset_rgb, offset_gray);	
	}

	for (int i = 0; i < stream_count; i++) {
		err = hipStreamSynchronize(streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}
	}

	//execute kernel for sobel
	for (int i = 0; i < stream_count; i++) {
		int offset_sobel = i * stream_size;

		//sobelFilterKernelStreams<<<blocks_per_grid, threads_per_block, 0, streams[i]>>>(d_image_width, d_image_height, d_dest_image, d_dest_image_sobel, offset_sobel);
		sobelFilterKernelTiledStreams<<<blocks_per_grid_tiled, threads_per_block_tiled, 0, streams[i] >> >(d_image_width, d_image_height, d_dest_image, d_dest_image_sobel, offset_sobel);
	}

	for (int i = 0; i < stream_count; i++) {
		err = hipStreamSynchronize(streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}
	}

	// Save grayscale data
	for (int i = 0; i < stream_count; i++) {
		int offset = i * stream_size;
		//printf("offset: %d\n", offset);
		err = hipMemcpyAsync(&dest_image[offset], &d_dest_image_sobel[offset], stream_size_gray, hipMemcpyDeviceToHost, streams[i]);
		if (err != hipSuccess) {
			printf("%s in %s at line %d\n",
				hipGetErrorString(err), __FILE__, __LINE__);
			exit(EXIT_FAILURE);
		}

	}

	hipHostFree(d_image_width);
	hipHostFree(d_image_height);
	hipHostFree(d_src_image);
	hipHostFree(d_dest_image);
	hipHostFree(d_dest_image_sobel);

	for (int i = 0; i < stream_count; i++) {
		hipStreamDestroy(streams[i]);
	}
	
};