#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void matMultCuda(float *cu_C, float *cu_A, float *cu_B, unsigned int n) {

	int i = threadIdx.x;
	cu_C[i] += cu_A[i] * cu_B[i];

};

void matMultHost(float* h_A, float* h_B, float* h_C, int n) // n = m
{
	for (int row = 0; row < n; ++row)
	{
		for (int col = 0; col < n; ++col)
		{
			for (int elem = 0; elem < n; ++elem)
			{
				h_C[row * n + col] += h_A[row * n + elem] * h_B[elem * n + col];
			}
		}
	}
}

void printMatrixHost(float* h_C, int n)
{
	for (int row = 0; row < n; ++row)
	{
		for (int col = 0; col < n; ++col)
		{
			printf("%f ", h_C[row * n + col]);
		}
		printf("\n");
	}
}

int main()
{
	unsigned int const n = 3;
	float *d_A, *d_B, *d_C;

	float h_A[] = { 
		1,1,1,
		2,2,2,
		3,3,3 };

	float h_B[] = { 
		1,1,1,
		2,2,2,
		3,3,3 };

	float h_C[n * n] = {};
	matMultHost(h_A, h_B, h_C, n);
	printMatrixHost(h_C, n);


	unsigned int memorySize = (n * n) * sizeof(float);

	hipError_t err = hipSuccess;

	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_A, memorySize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_A, h_A, memorySize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_B, memorySize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, memorySize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_C, memorySize);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_C, h_C, memorySize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	int num_blocks = 1;
	int num_threads = 9;
	matMultCuda <<<num_blocks, num_threads>>> (d_C, d_A, d_B, n);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(h_C, d_C, memorySize, hipMemcpyDeviceToHost);

	printMatrixHost(h_C, n);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<1, size>>> (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
