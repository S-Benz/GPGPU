
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <cstdlib>
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void mathMultCuda(float *cu_C, float *cu_A, float *cu_B, unsigned int n) {

	int i = threadIdx.x;
	cu_C[i] += cu_A[i] * cu_B[i];

};

int main()
{

	unsigned int const matrix_element_size = 9;
	float *d_A, *d_B, *d_C;

	float h_A[] = { 1,1,1,
			2,2,2,
			3,3,3 };

	float h_B[] = { 1,1,1,
		2,2,2,
		3,3,3 };

	float h_C[matrix_element_size] = {};


	unsigned int size = matrix_element_size * sizeof(float);

	hipError_t err = hipSuccess;

	err = hipSetDevice(0);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_A, size);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_B, size);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMalloc((void **)&d_C, size);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n",
			hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}

	int num_blocks = 3;
	int num_threads = 3;

	mathMultCuda<<<num_blocks, num_threads>>>(d_C, d_A, d_B, matrix_element_size);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", err);
	}

	err = hipMemcpy(h_C, d_C, matrix_element_size * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
